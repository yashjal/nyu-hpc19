
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void reduction(double* sum_ptr, const double* a, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i];
  *sum_ptr = sum;
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

// Warp divergence
__global__ void reduction_kernel0(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x %   2 == 0) smem[threadIdx.x] += smem[threadIdx.x + 1];
  __syncthreads();
  if (threadIdx.x %   4 == 0) smem[threadIdx.x] += smem[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x %   8 == 0) smem[threadIdx.x] += smem[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x %  16 == 0) smem[threadIdx.x] += smem[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x %  32 == 0) smem[threadIdx.x] += smem[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x %  64 == 0) smem[threadIdx.x] += smem[threadIdx.x + 32];
  __syncthreads();
  if (threadIdx.x % 128 == 0) smem[threadIdx.x] += smem[threadIdx.x + 64];
  __syncthreads();
  if (threadIdx.x % 256 == 0) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x % 512 == 0) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[threadIdx.x] + smem[threadIdx.x + 512];
}

// Shared memory bank conflicts
__global__ void reduction_kernel1(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x *   2] += smem[threadIdx.x *   2 +   1];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x *   4] += smem[threadIdx.x *   4 +   2];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x *   8] += smem[threadIdx.x *   8 +   4];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x *  16] += smem[threadIdx.x *  16 +   8];
  __syncthreads();
  if (threadIdx.x <  32) smem[threadIdx.x *  32] += smem[threadIdx.x *  32 +  16];
  __syncwarp();
  if (threadIdx.x <  16) smem[threadIdx.x *  64] += smem[threadIdx.x *  64 +  32];
  __syncwarp();
  if (threadIdx.x <   8) smem[threadIdx.x * 128] += smem[threadIdx.x * 128 +  64];
  __syncwarp();
  if (threadIdx.x <   4) smem[threadIdx.x * 256] += smem[threadIdx.x * 256 + 128];
  __syncwarp();
  if (threadIdx.x <   2) smem[threadIdx.x * 512] += smem[threadIdx.x * 512 + 256];
  __syncwarp();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[512];
}

__global__ void reduction_kernel2(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void vec_mult_kernel(double* c, const double* a, const double* b, long N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) c[idx] = a[idx] * b[idx];
}


int main() {
  long N = (1UL<<25);

  double *x;
  double *y;
  double *z;
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&z, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = 1.0;
    y[i] = 1.0;
  }

  double sum_ref, sum;
  double tt = omp_get_wtime();
  for (long i=0; i<N;i++) {
    z[i] = x[i] * y[i];
  }
  reduction(&sum_ref, z, N);
  printf("CPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *y_d, *z_d, *c_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&y_d, N*sizeof(double));
  hipMalloc(&z_d, N*sizeof(double));
  hipMalloc(&c_d, ((N+BLOCK_SIZE-1)/BLOCK_SIZE)*sizeof(double));

  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();

  vec_mult_kernel<<<N/BLOCK_SIZE+1,BLOCK_SIZE>>>(z_d, y_d, x_d, N);
  hipDeviceSynchronize();
  double* sum_d = c_d;
  long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
  reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d, z_d, N);
  while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d + Nb, sum_d, N);
    sum_d += Nb;
  }

  hipMemcpyAsync(&sum, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(sum-sum_ref));
  printf("sum %f\n", sum);
  printf("sum_ref %f\n", sum_ref);

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(c_d);
  hipHostFree(x);
  hipHostFree(y);
  hipHostFree(z);

  return 0;
}
